#include "hip/hip_runtime.h"
#include "KMeans_CUDA.cuh"

KMeans_CUDA::KMeans_CUDA(int nClusters, int maxIter, uint seed) : KMeans(nClusters, maxIter, seed) {}

// flattened device arrays
__device__ el_type *d_points, *d_centroids, *d_centroidsNew;
__device__ unsigned int *d_pointsPerCluster, *d_labels;

const KMeans *KMeans_CUDA::fit(const matrix &points) {
    const auto &points_n = points[0].size();
    const auto &dimension_n = points.size();
    const auto &cluster_n = getNClusters();

    // 1. select random observation as starting centroids
    matrix centroids{dimension_n};

    std::mt19937 mt(getSeed());
    std::uniform_real_distribution<double> dist(0, points_n);

    for (int i=0; i<cluster_n; ++i) {
        auto rnd_i = static_cast<uint>(dist(mt));
        for (int ii=0; ii<dimension_n; ++ii) {
            centroids[ii].push_back(points[ii][rnd_i]);
        }
    }

    // points
    el_type *dev_points;
    {
        // allocate device memory
        size_t sz = points_n * sizeof(el_type);
        CUDA_CHECK_RETURN(hipMalloc((void **) &(dev_points), dimension_n * sz));
        // copy arrays from host to device (flattened)
        CUDA_CHECK_RETURN(hipMemcpy(dev_points, points[0].data(), sz, hipMemcpyHostToDevice));
        // copy second dimension array with offset
        CUDA_CHECK_RETURN(hipMemcpy(dev_points + points_n, points[1].data(), sz, hipMemcpyHostToDevice));
        // copy host struct to device
        CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_points), &dev_points, sizeof(el_type *)));
    }

    // centroids
    el_type *dev_centroids;
    {
        size_t sz = cluster_n * sizeof(el_type);
        CUDA_CHECK_RETURN(hipMalloc((void **) &(dev_centroids), dimension_n * sz));
        CUDA_CHECK_RETURN(hipMemcpy(dev_centroids, centroids[0].data(), sz, hipMemcpyHostToDevice));
        CUDA_CHECK_RETURN(hipMemcpy(dev_centroids + cluster_n, centroids[1].data(), sz, hipMemcpyHostToDevice));
        CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_centroids), &dev_centroids, sizeof(el_type *)));
    }

    // centroidsNew
    el_type *dev_centroidsNew;
    {
        size_t sz = cluster_n * sizeof(el_type);
        CUDA_CHECK_RETURN(hipMalloc((void **) &(dev_centroidsNew), dimension_n * sz));
        // init to 0
        CUDA_CHECK_RETURN(hipMemset(dev_centroidsNew, 0, dimension_n * sz));
        CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_centroidsNew), &dev_centroidsNew, sizeof(el_type *)));
    }

    // pointsPerCluster
    unsigned int *dev_pointsPerCluster;
    {
        size_t sz = cluster_n * sizeof(unsigned int);
        CUDA_CHECK_RETURN(hipMalloc((void **) &dev_pointsPerCluster, sz));
        CUDA_CHECK_RETURN(hipMemset(dev_pointsPerCluster, 0, sz));
        CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_pointsPerCluster), &dev_pointsPerCluster, sizeof(unsigned int *)));
    }

    // labels
    unsigned int *dev_labels;
    {
        size_t sz = points_n * sizeof(unsigned int);
        CUDA_CHECK_RETURN(hipMalloc((void **) &dev_labels, sz));
        CUDA_CHECK_RETURN(hipMemset(dev_labels, 0, sz));
        CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_labels), &dev_labels, sizeof(unsigned int *)));
    }

    {
        unsigned int threadsPerBlock = 128;
        unsigned int blocks = std::ceil((float) points_n / threadsPerBlock);

        size_t cN_sz = dimension_n * cluster_n;
        size_t cN_sz_bytes = cN_sz * sizeof(el_type);
        size_t pPC_sz_bytes = cluster_n * sizeof(unsigned int);
        int maxIter = getMaxIter(), nIter = 0;
        while (nIter < maxIter) {
            ++nIter;

            assignPointsToCentroids<<<blocks, threadsPerBlock, dimension_n*cluster_n*sizeof(el_type)>>>(dimension_n, points_n, cluster_n);
            hipDeviceSynchronize();

            calculateNewCentroids<<<1, cluster_n>>>(dimension_n, points_n, cluster_n);
            hipDeviceSynchronize();

            CUDA_CHECK_RETURN(hipMemset(dev_centroidsNew, 0, cN_sz_bytes));
            CUDA_CHECK_RETURN(hipMemset(dev_pointsPerCluster, 0, pPC_sz_bytes));
        }

        // copy data back to host
        unsigned int h_labels[points_n];
        CUDA_CHECK_RETURN(hipMemcpy(&h_labels, dev_labels, points_n * sizeof(unsigned int), hipMemcpyDeviceToHost));

        el_type h_centroids[dimension_n*cluster_n];
        CUDA_CHECK_RETURN(hipMemcpy(&h_centroids, dev_centroids, dimension_n*cluster_n * sizeof(el_type), hipMemcpyDeviceToHost));
        for (size_t d_i = 0; d_i < dimension_n; d_i++) {
            for (size_t i = 0; i < cluster_n; i++) {
                centroids[d_i][i] = h_centroids[d_i * cluster_n + i];
            }
        }

        clusterCenters = centroids;
        this->labels.insert(this->labels.begin(), h_labels, h_labels+points_n);
        this->nIter = nIter;
    }

    // free memory
    CUDA_CHECK_RETURN(hipFree(dev_points));
    CUDA_CHECK_RETURN(hipFree(dev_centroids));
    CUDA_CHECK_RETURN(hipFree(dev_centroidsNew));
    CUDA_CHECK_RETURN(hipFree(dev_pointsPerCluster));
    CUDA_CHECK_RETURN(hipFree(dev_labels));

    return this;
}

__global__ void
assignPointsToCentroids(size_t dimension_n, size_t points_n, size_t cluster_n) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // load centroids in shared memory
    extern __shared__ el_type s_centroids[];
    for (int i = threadIdx.x; i < cluster_n; i += blockDim.x) {
        for (int j = 0; j < dimension_n; j++) {
            s_centroids[cluster_n * j + i] = d_centroids[cluster_n * j + i];
        }
    }
    __syncthreads();

    if (idx < points_n) {
        // 2. compute the distance between centroids and observation
        unsigned int label = 0; // index of the centroid closer to p
        {
            double min = -1; // minimum distance between p and a centroid
            for (size_t j = 0; j < cluster_n; j++) {
                // euclidean distance (norm2) between centroid and point
                // sqrt is unnecessary since it's a monotone function
                el_type d = 0;
                for (size_t d_i = 0; d_i < dimension_n; ++d_i) {
                    // d_i * points_n is the coordinate offset since array is flat
                    el_type sub = d_points[d_i * points_n + idx] - s_centroids[d_i * cluster_n + j];
                    d += sub * sub;
                }
                if (d < min || min == -1) {
                    min = d;
                    label = j;
                }
            }
        }
        // update our point label (no sync because every thread writes its index)
        d_labels[idx] = label;

        // 3. assign each observation to a centroid based on their distance
        // atomically increment the global counter array
        atomicAdd(&(d_pointsPerCluster[label]), 1);

        // 4. compute new centroids (mean of observations in a cluster)
        // atomically sum thread point at centroid index
        // the actual mean will be done in another kernel since we need to sync
        // every thread of the grid to get final pointsPerCluster values
        for (size_t d_i = 0; d_i < dimension_n; d_i++) {
            atomicAdd(&(d_centroidsNew[d_i * cluster_n + label]),
                            d_points[d_i * points_n + idx]);
        }
    }
}

__global__ void
calculateNewCentroids(size_t dimension_n, size_t points_n, size_t cluster_n) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // calculate new centroids
    if(idx < cluster_n){
        for (size_t d_i=0; d_i<dimension_n; ++d_i) {
            d_centroids[d_i * cluster_n + idx] = d_centroidsNew[d_i * cluster_n + idx] / d_pointsPerCluster[idx];
        }
    }
}
